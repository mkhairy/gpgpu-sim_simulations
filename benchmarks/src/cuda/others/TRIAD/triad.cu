#include <iostream>
#include <cstdio>
using namespace std;
#include <hip/hip_runtime.h>
#define TIMES 24


#include<sm_35_intrinsics.h>

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////HELP FUNCTIONS/////////////////////////////////////////////////
void RandomInit(float* data, int n)
{
    for (int i=0; i<n; i++)
	{
        data[i] = rand() / (float)RAND_MAX;
	}
}

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling cudaGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////_VECTOR_ADDITION_///////////////////////////////////////////////////////
__global__ void triad(const float* A, const float* B, float* C, int scale)

{

    int i = blockDim.x * blockIdx.x + threadIdx.x;

        C[i] = A[i] + scale * B[i];

}


// Host code
void VectorAddition(int N, int threadsPerBlock, int compute, int scale)
{
	cout<<"Vector Addition for input size "<<N<<" :\n";
	// Variables
	float* h_A;
	float* h_B;
	float* h_C;
        float* h_D;
	float* d_A;
	float* d_B;
	float* d_C;
        float* d_D;
	float total_time=0;
    size_t size = N * sizeof(float) * scale;

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    h_D = (float*)malloc(size);
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        cout<<"deviceIndex"<<deviceIndex<<", SM = "<<deviceProperties.major<<"."<<deviceProperties.minor<<"L1 cache: " <<deviceProperties.globalL1CacheSupported<<endl;
        if (deviceProperties.major >= compute
            && deviceProperties.minor >= 0)
        {
            cout<<"Set device to "<<deviceIndex<<endl;
            hipSetDevice(deviceIndex);
        }
    }

    // Allocate vectors in device memory
    checkCudaErrors( hipMalloc((void**)&d_A, size) );
    checkCudaErrors( hipMalloc((void**)&d_B, size) );
    checkCudaErrors( hipMalloc((void**)&d_C, size) );
    checkCudaErrors( hipMalloc((void**)&d_D, size) );

    // Copy vectors from host memory to device memory
    checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );
 
    checkCudaErrors(hipDeviceSynchronize());
    // Invoke kernel
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	for (int i = 0; i < 1; i++) {

    triad<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, scale);

	checkCudaErrors(hipDeviceSynchronize());
	}

	double dSeconds = total_time/((double)TIMES * 1000);
	double dNumOps = N;
	double gflops = 1.0e-9 * dNumOps/dSeconds;
	cout<<"Time = "<<dSeconds*1.0e3<< "msec"<<endl<<"gflops = "<<gflops<<endl;

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    
    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }

        // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    hipDeviceReset();

	if(i == N)
		cout<<"SUCCSESS"<<endl;
	else 
		cout<<"FAILED"<<endl;   
}
//////////////////////////////////////////////////////
int main(int argc,char *argv[])
{ 
  if(argc < 4)
     printf("Unsuffcient number of arguments!\n");
else
	{
		VectorAddition(atoi(argv[1]), atoi(argv[2]), atoi(argv[3]), atoi(argv[4]));
	}
}
